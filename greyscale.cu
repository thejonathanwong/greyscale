#include "hip/hip_runtime.h"


#include <time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

// STUDENTS: be sure to set the single define at the top of this file, 
// depending on which machines you are running on.
#include "im1.h"



// handy error macro:
#define GPU_CHECKERROR( err ) (gpuCheckError( err, __FILE__, __LINE__ ))
static void gpuCheckError( hipError_t err,
                          const char *file,
                          int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
               file, line );
        exit( EXIT_FAILURE );
    }
}

int gpuDevSel() { //method to select device with largest number of max threads in a block
		// returns that maxThread number

	int dev_count;
	GPU_CHECKERROR( hipGetDeviceCount( &dev_count ) );


	hipDeviceProp_t dev_prop;
	unsigned int  maxThreads = 0;
	unsigned int devNum = 0;
	for(int i = 0; i < dev_count; i++) { //loops through CUDA devices
		GPU_CHECKERROR( hipGetDeviceProperties( &dev_prop, i ) );
		if(dev_prop.maxThreadsPerBlock > maxThreads) { //sets devNum to device with
			maxThreads =(unsigned int) dev_prop.maxThreadsPerBlock; //highest num threads per 
			devNum = i;				//block
		}
	}
	GPU_CHECKERROR( hipSetDevice(devNum) ); //selects CUDA device

	return maxThreads;

}

__global__ void gpuCalLum(float * array, int width, int height) {
	//origin is top left pixel of the block
	int originX = blockDim.x * blockIdx.x; 
	int originY = blockDim.y * blockIdx.y;

	//indexZ is number of pixels right or down of the origin in the block
	int indexX = threadIdx.x;
	int indexY = threadIdx.y;

	//maps the current thread/pixel to a position in the 1D array of RGB values
	int index = 3*((originX+indexX)+(originY + indexY)*width);
/*
	float red = **(array + index)*0.2126f; 
	float green = **(array + index + 1)*0.7152f;
	float blue = **(array + index + 1)*0.0722f;
*/

	if(((originY + indexY) < height) &&  ((originX + indexX) < width)) { 
		float L = 0.2126f*array[index]+
			  0.7152f*array[index+1]+
			  0.0722f*array[index+2];
		
		*(array+index)= L;
		*(array+index+1)= L;
		*(array+index+2)= L;
	}	

}


//
// your __global__ kernel can go here, if you want:
//


int main (int argc, char *argv[])
{
 

	clock_t timer1, timer2;
    printf("reading openEXR file %s\n", argv[1]);
        
    int w, h;   // the width & height of the image, used frequently!


    // First, convert the openEXR file into a form we can use on the CPU
    // and the GPU: a flat array of floats:
    // This makes an array h*w*sizeof(float)*3, with sequential r/g/b indices
    // don't forget to free it at the end


    timer1 = clock();
    float *h_imageArray;
    readOpenEXRFile (argv[1], &h_imageArray, w, h);

    // 
    // serial code: saves the image in "hw1_serial.exr"
    //

    // for every pixel in p, get it's Rgba structure, and convert the
    // red/green/blue values there to luminance L, effectively converting
    // it to greyscale:

    for (int y = 0; y < h; ++y) {
        for (int x = 0; x < w; ++x) {
            
            unsigned int idx = ((y * w) + x) * 3;
            
            float L = 0.2126f*h_imageArray[idx] + 
                      0.7152f*h_imageArray[idx+1] + 
                      0.0722f*h_imageArray[idx+2];

            h_imageArray[idx] = L;
            h_imageArray[idx+1] = L;
            h_imageArray[idx+2] = L;

       }
    }
    
    printf("writing output image hw1_serial.exr\n");
    writeOpenEXRFile ("hw1_serial.exr", h_imageArray, w, h);
    free(h_imageArray); // make sure you free it: if you use this variable
                        // again, readOpenEXRFile will allocate more memory

	timer1 = clock() - timer1;

    //
    // Now the GPU version: it will save whatever is in h_imageArray
    // to the file "hw1_gpu.exr"
    //
    
	timer2 = clock();
    // read the file again - the file read allocates memory for h_imageArray:
    readOpenEXRFile (argv[1], &h_imageArray, w, h);



    // at this point, h_imageArray has sequenial floats for red, green , and
    // blue for each pixel: r,g,b,r,g,b,r,g,b,r,g,b. You need to copy
    // this array to GPU global memory, and have one thread per pixel compute
    // the luminance value, with which you will overwrite each r,g,b, triple.

    //
    // process it on the GPU: 1) copy it to device memory, 2) process
    // it with a 2d grid of 2d blocks, with each thread assigned to a 
    // pixel. then 3) copy it back.
    //
	unsigned int numPixels = w * h;
	unsigned int arraySize = 3 * numPixels;
	unsigned int maxThreads;
    	maxThreads = gpuDevSel();

	unsigned int threadDim = sqrt(maxThreads);
//	printf("threadDim =%d\n", threadDim);
	//unsigned int numBlocks =  ceil( numPixels/ (float) maxThreads );
	int gridWidth = ceil( (float) w/threadDim );
	int gridHeight = ceil( (float) h/threadDim );
//	printf("gridWidth = %d, gridHeight = %d\n", gridWidth, gridHeight);
	dim3 grid(gridWidth,gridHeight);
	dim3 threads(threadDim, threadDim);	

//	printf("width = %d, height = %d\n", w, h);


	//creates and allocates memory to the device array
	//copies array from host to device
	float * d_imageArray;
	GPU_CHECKERROR( hipMalloc((void **) &d_imageArray, arraySize*sizeof(float)) ) ; 
	GPU_CHECKERROR( hipMemcpy((void *) d_imageArray, (void *) h_imageArray, arraySize*sizeof(float), 
			hipMemcpyHostToDevice));

	gpuCalLum<<<grid, threads>>>(d_imageArray,w,h);


    //
    // Your memory copy, & kernel launch code goes here:
    //


//	copies array back from device to host and frees device array
	GPU_CHECKERROR( hipMemcpy((void *) h_imageArray, (void *) d_imageArray, arraySize*sizeof(float), hipMemcpyDeviceToHost));
	GPU_CHECKERROR( hipFree((void *) d_imageArray) );


    // All your work is done. Here we assume that you have copied the 
    // processed image data back, frmm the device to the host, into the
    // original host array h_imageArray. You can do it some other way,
    // this is just a suggestion
    
    printf("writing output image hw1_gpu.exr\n");
    writeOpenEXRFile ("hw1_gpu.exr", h_imageArray, w, h);
    free (h_imageArray);
    timer2 = clock() - timer2;
	printf("This was calculated serially in %f seconds.\n", ((float)timer1)/CLOCKS_PER_SEC);
	printf("This was calculated in parallel in %f seconds.\n", ((float)timer2)/CLOCKS_PER_SEC);

    printf("done.\n");

    return 0;
}


